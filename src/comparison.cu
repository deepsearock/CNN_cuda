#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>

// Macro for error checking
#define CUDA_CHECK(call) {                                              \
    hipError_t err = call;                                             \
    if(err != hipSuccess) {                                             \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,  \
                hipGetErrorString(err));                               \
        exit(EXIT_FAILURE);                                             \
    }                                                                   \
}

// optimized kernel
texture<int, hipTextureType2D, hipReadModeElementType> texRef;

__global__ void convolution2DKernel(int *output, int imageWidth, int imageHeight, int maskWidth, int maskRadius) {
    extern __shared__ int sharedMem[];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;
    
    int sharedWidth = blockDim.x + 2 * maskRadius;
    int shared_x = tx + maskRadius;
    int shared_y = ty + maskRadius;
    
    if (row < imageHeight && col < imageWidth)
        sharedMem[shared_y * sharedWidth + shared_x] = tex2D(texRef, col, row);
    else
        sharedMem[shared_y * sharedWidth + shared_x] = 0;
    
    if (tx < maskRadius) {
        int halo_col = col - maskRadius;
        int value = (halo_col >= 0 && row < imageHeight) ? tex2D(texRef, halo_col, row) : 0;
        sharedMem[shared_y * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius) {
        int halo_col = col + maskRadius;
        int value = (halo_col < imageWidth && row < imageHeight) ? tex2D(texRef, halo_col, row) : 0;
        sharedMem[shared_y * sharedWidth + shared_x + maskRadius] = value;
    }
    if (ty < maskRadius) {
        int halo_row = row - maskRadius;
        int value = (halo_row >= 0 && col < imageWidth) ? tex2D(texRef, col, halo_row) : 0;
        sharedMem[ty * sharedWidth + shared_x] = value;
    }
    if (ty >= blockDim.y - maskRadius) {
        int halo_row = row + maskRadius;
        int value = (halo_row < imageHeight && col < imageWidth) ? tex2D(texRef, col, halo_row) : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + shared_x] = value;
    }
    if (tx < maskRadius && ty < maskRadius) {
        int halo_col = col - maskRadius;
        int halo_row = row - maskRadius;
        int value = (halo_col >= 0 && halo_row >= 0) ? tex2D(texRef, halo_col, halo_row) : 0;
        sharedMem[ty * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius && ty < maskRadius) {
        int halo_col = col + maskRadius;
        int halo_row = row - maskRadius;
        int value = (halo_col < imageWidth && halo_row >= 0) ? tex2D(texRef, halo_col, halo_row) : 0;
        sharedMem[ty * sharedWidth + shared_x + maskRadius] = value;
    }
    if (tx < maskRadius && ty >= blockDim.y - maskRadius) {
        int halo_col = col - maskRadius;
        int halo_row = row + maskRadius;
        int value = (halo_col >= 0 && halo_row < imageHeight) ? tex2D(texRef, halo_col, halo_row) : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius && ty >= blockDim.y - maskRadius) {
        int halo_col = col + maskRadius;
        int halo_row = row + maskRadius;
        int value = (halo_col < imageWidth && halo_row < imageHeight) ? tex2D(texRef, halo_col, halo_row) : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + shared_x + maskRadius] = value;
    }
    
    __syncthreads();
    
    if (row < imageHeight && col < imageWidth) {
        int output_value = 0;
        for (int i = -maskRadius; i <= maskRadius; i++) {
            for (int j = -maskRadius; j <= maskRadius; j++) {
                int image_value = sharedMem[(shared_y + i) * sharedWidth + (shared_x + j)];
                output_value += image_value;
            }
        }
        output_value /= ((2 * maskRadius + 1) * (2 * maskRadius + 1));
        output[row * imageWidth + col] = output_value;
    }
}

// naive kernel
__global__ void naiveConvolutionKernel(int *input, int *output, int imageWidth, int imageHeight, int maskWidth, int maskRadius) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < imageHeight && col < imageWidth) {
        int sum = 0;
        for (int i = -maskRadius; i <= maskRadius; i++) {
            for (int j = -maskRadius; j <= maskRadius; j++) {
                int r = row + i;
                int c = col + j;
                int value = 0;
                if (r >= 0 && r < imageHeight && c >= 0 && c < imageWidth)
                    value = input[r * imageWidth + c];
                sum += value;
            }
        }
        output[row * imageWidth + col] = sum / ((2 * maskRadius + 1) * (2 * maskRadius + 1));
    }
}

// shared memory kernel
__global__ void sharedMemoryConvolutionKernel(int *input, int *output, int imageWidth, int imageHeight, int maskWidth, int maskRadius) {
    extern __shared__ int sharedMem[];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;
    
    int sharedWidth = blockDim.x + 2 * maskRadius;
    int shared_x = tx + maskRadius;
    int shared_y = ty + maskRadius;
    
    if (row < imageHeight && col < imageWidth)
        sharedMem[shared_y * sharedWidth + shared_x] = input[row * imageWidth + col];
    else
        sharedMem[shared_y * sharedWidth + shared_x] = 0;
    
    if (tx < maskRadius) {
        int halo_col = col - maskRadius;
        int value = (halo_col >= 0 && row < imageHeight) ? input[row * imageWidth + halo_col] : 0;
        sharedMem[shared_y * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius) {
        int halo_col = col + maskRadius;
        int value = (halo_col < imageWidth && row < imageHeight) ? input[row * imageWidth + halo_col] : 0;
        sharedMem[shared_y * sharedWidth + shared_x + maskRadius] = value;
    }
    if (ty < maskRadius) {
        int halo_row = row - maskRadius;
        int value = (halo_row >= 0 && col < imageWidth) ? input[halo_row * imageWidth + col] : 0;
        sharedMem[ty * sharedWidth + shared_x] = value;
    }

    if (ty >= blockDim.y - maskRadius) {
        int halo_row = row + maskRadius;
        int value = (halo_row < imageHeight && col < imageWidth) ? input[halo_row * imageWidth + col] : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + shared_x] = value;
    }

    if (tx < maskRadius && ty < maskRadius) {
        int halo_col = col - maskRadius;
        int halo_row = row - maskRadius;
        int value = (halo_col >= 0 && halo_row >= 0) ? input[halo_row * imageWidth + halo_col] : 0;
        sharedMem[ty * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius && ty < maskRadius) {
        int halo_col = col + maskRadius;
        int halo_row = row - maskRadius;
        int value = (halo_col < imageWidth && halo_row >= 0) ? input[halo_row * imageWidth + halo_col] : 0;
        sharedMem[ty * sharedWidth + shared_x + maskRadius] = value;
    }
    if (tx < maskRadius && ty >= blockDim.y - maskRadius) {
        int halo_col = col - maskRadius;
        int halo_row = row + maskRadius;
        int value = (halo_col >= 0 && halo_row < imageHeight) ? input[halo_row * imageWidth + halo_col] : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius && ty >= blockDim.y - maskRadius) {
        int halo_col = col + maskRadius;
        int halo_row = row + maskRadius;
        int value = (halo_col < imageWidth && halo_row < imageHeight) ? input[halo_row * imageWidth + halo_col] : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + shared_x + maskRadius] = value;
    }
    
    __syncthreads();
    
    if (row < imageHeight && col < imageWidth) {
        int sum = 0;
        for (int i = -maskRadius; i <= maskRadius; i++) {
            for (int j = -maskRadius; j <= maskRadius; j++) {
                sum += sharedMem[(shared_y + i) * sharedWidth + (shared_x + j)];
            }
        }
        output[row * imageWidth + col] = sum / ((2 * maskRadius + 1) * (2 * maskRadius + 1));
    }
}

//cpu
void convolution2D_CPU(const int *h_image, int *h_output, int imageWidth, int imageHeight, int maskWidth) {
    int maskRadius = maskWidth / 2;
    for (int row = 0; row < imageHeight; row++) {
        for (int col = 0; col < imageWidth; col++) {
            int sum = 0;
            for (int i = -maskRadius; i <= maskRadius; i++) {
                for (int j = -maskRadius; j <= maskRadius; j++) {
                    int curRow = row + i;
                    int curCol = col + j;
                    int value = 0;
                    if (curRow >= 0 && curRow < imageHeight && curCol >= 0 && curCol < imageWidth)
                        value = h_image[curRow * imageWidth + curCol];
                    sum += value;
                }
            }
            h_output[row * imageWidth + col] = sum / ((2 * maskRadius + 1) * (2 * maskRadius + 1));
        }
    }
}

int main(int argc, char **argv) {
    // defualt
    int dimX = 512, dimY = 512, dimK = 3;
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-i") == 0)
            dimX = atoi(argv[++i]);
        else if (strcmp(argv[i], "-j") == 0)
            dimY = atoi(argv[++i]);
        else if (strcmp(argv[i], "-k") == 0)
            dimK = atoi(argv[++i]);
    }
    
    int imageSize = dimX * dimY;
    
    // allocate host memory
    int *h_image         = (int *)malloc(sizeof(int) * imageSize);
    int *h_output_gpu      = (int *)malloc(sizeof(int) * imageSize); 
    int *h_output_naive    = (int *)malloc(sizeof(int) * imageSize);
    int *h_output_shared   = (int *)malloc(sizeof(int) * imageSize);
    int *h_output_cpu      = (int *)malloc(sizeof(int) * imageSize); 
    
    // rand
    for (int i = 0; i < imageSize; i++) {
        h_image[i] = rand() % 16;
    }
    
    int maskWidth = dimK;
    int maskRadius = maskWidth / 2;
    
// texture memory kernel
    hipArray *d_imageArray;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
    CUDA_CHECK(hipMallocArray(&d_imageArray, &channelDesc, dimX, dimY));
    
    CUDA_CHECK(hipMemcpy2DToArray(d_imageArray, 0, 0, h_image, dimX * sizeof(int),
                                   dimX * sizeof(int), dimY, hipMemcpyHostToDevice));
    
    texRef.addressMode[0] = hipAddressModeClamp;
    texRef.addressMode[1] = hipAddressModeClamp;
    texRef.filterMode     = hipFilterModePoint;
    texRef.normalized     = false;
    CUDA_CHECK(hipBindTextureToArray(texRef, d_imageArray, channelDesc));
    
    int *d_output;
    CUDA_CHECK(hipMalloc(&d_output, sizeof(int) * imageSize));
    
    dim3 blockDim(16, 16);
    dim3 gridDim((dimX + blockDim.x - 1) / blockDim.x, (dimY + blockDim.y - 1) / blockDim.y);
    int sharedMemSize = (blockDim.x + 2 * maskRadius) * (blockDim.y + 2 * maskRadius) * sizeof(int);
    
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    CUDA_CHECK(hipEventRecord(start, 0));
    convolution2DKernel<<<gridDim, blockDim, sharedMemSize>>>(d_output, dimX, dimY, maskWidth, maskRadius);
    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));
    
    float elapsedTime_gpu;
    CUDA_CHECK(hipEventElapsedTime(&elapsedTime_gpu, start, stop));
    CUDA_CHECK(hipGetLastError());
    
    CUDA_CHECK(hipMemcpy(h_output_gpu, d_output, sizeof(int) * imageSize, hipMemcpyDeviceToHost));
    
    CUDA_CHECK(hipUnbindTexture(texRef));
    CUDA_CHECK(hipFreeArray(d_imageArray));
    CUDA_CHECK(hipFree(d_output));
    
// naive kernel
    int *d_input, *d_output_naive;
    CUDA_CHECK(hipMalloc(&d_input, sizeof(int) * imageSize));
    CUDA_CHECK(hipMalloc(&d_output_naive, sizeof(int) * imageSize));
    
    CUDA_CHECK(hipMemcpy(d_input, h_image, sizeof(int) * imageSize, hipMemcpyHostToDevice));
    
    hipEvent_t startNaive, stopNaive;
    CUDA_CHECK(hipEventCreate(&startNaive));
    CUDA_CHECK(hipEventCreate(&stopNaive));
    
    CUDA_CHECK(hipEventRecord(startNaive, 0));
    naiveConvolutionKernel<<<gridDim, blockDim>>>(d_input, d_output_naive, dimX, dimY, maskWidth, maskRadius);
    CUDA_CHECK(hipEventRecord(stopNaive, 0));
    CUDA_CHECK(hipEventSynchronize(stopNaive));
    
    float elapsedTime_naive;
    CUDA_CHECK(hipEventElapsedTime(&elapsedTime_naive, startNaive, stopNaive));
    CUDA_CHECK(hipGetLastError());
    
    CUDA_CHECK(hipMemcpy(h_output_naive, d_output_naive, sizeof(int) * imageSize, hipMemcpyDeviceToHost));
    
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output_naive));
    
//shared memory kernel
    int *d_input_shared, *d_output_shared;
    CUDA_CHECK(hipMalloc(&d_input_shared, sizeof(int) * imageSize));
    CUDA_CHECK(hipMalloc(&d_output_shared, sizeof(int) * imageSize));
    
    CUDA_CHECK(hipMemcpy(d_input_shared, h_image, sizeof(int) * imageSize, hipMemcpyHostToDevice));
    
    hipEvent_t startShared, stopShared;
    CUDA_CHECK(hipEventCreate(&startShared));
    CUDA_CHECK(hipEventCreate(&stopShared));
    
    CUDA_CHECK(hipEventRecord(startShared, 0));
    sharedMemoryConvolutionKernel<<<gridDim, blockDim, sharedMemSize>>>(d_input_shared, d_output_shared, dimX, dimY, maskWidth, maskRadius);
    CUDA_CHECK(hipEventRecord(stopShared, 0));
    CUDA_CHECK(hipEventSynchronize(stopShared));
    
    float elapsedTime_shared;
    CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared, startShared, stopShared));
    CUDA_CHECK(hipGetLastError());
    
    CUDA_CHECK(hipMemcpy(h_output_shared, d_output_shared, sizeof(int) * imageSize, hipMemcpyDeviceToHost));
    
    CUDA_CHECK(hipFree(d_input_shared));
    CUDA_CHECK(hipFree(d_output_shared));
    
    auto cpu_start = std::chrono::high_resolution_clock::now();
    convolution2D_CPU(h_image, h_output_cpu, dimX, dimY, maskWidth);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsedTime_cpu = cpu_end - cpu_start;
    

    int errorCount = 0;
    for (int i = 0; i < imageSize; i++) {
        if (abs(h_output_gpu[i] - h_output_cpu[i]) > 1e-5) {
            errorCount++;
            if (errorCount < 10) {
                printf("Optimized GPU mismatch at index %d: GPU = %d, CPU = %d\n", i, h_output_gpu[i], h_output_cpu[i]);
            }
        }
    }
    if (errorCount == 0)
        printf("Optimized GPU and CPU results match.\n");
    else
        printf("Total mismatches (Optimized GPU vs CPU): %d\n", errorCount);
    
    errorCount = 0;
    for (int i = 0; i < imageSize; i++) {
        if (abs(h_output_naive[i] - h_output_cpu[i]) > 1e-5) {
            errorCount++;
            if (errorCount < 10) {
                printf("Naive GPU mismatch at index %d: GPU = %d, CPU = %d\n", i, h_output_naive[i], h_output_cpu[i]);
            }
        }
    }
    if (errorCount == 0)
        printf("Naive GPU and CPU results match.\n");
    else
        printf("Total mismatches (Naive GPU vs CPU): %d\n", errorCount);
    
    errorCount = 0;
    for (int i = 0; i < imageSize; i++) {
        if (abs(h_output_shared[i] - h_output_cpu[i]) > 1e-5) {
            errorCount++;
            if (errorCount < 10) {
                printf("Shared Memory GPU mismatch at index %d: GPU = %d, CPU = %d\n", i, h_output_shared[i], h_output_cpu[i]);
            }
        }
    }
    if (errorCount == 0)
        printf("Shared Memory GPU and CPU results match.\n");
    else
        printf("Total mismatches (Shared Memory GPU vs CPU): %d\n", errorCount);
    

    double opsPerPixel = (maskWidth * maskWidth) + 1;
    double totalOps = imageSize * opsPerPixel;
    
    double seconds_gpu   = elapsedTime_gpu / 1000.0;
    double gflops_gpu    = (totalOps / seconds_gpu) / 1e9;
    
    double seconds_naive = elapsedTime_naive / 1000.0;
    double gflops_naive  = (totalOps / seconds_naive) / 1e9;
    
    double seconds_shared = elapsedTime_shared / 1000.0;
    double gflops_shared  = (totalOps / seconds_shared) / 1e9;
    
    double seconds_cpu   = elapsedTime_cpu.count() / 1000.0;
    double gflops_cpu    = (totalOps / seconds_cpu) / 1e9;
    
    printf("Optimized GPU kernel execution time: %f ms, Performance: %f GFLOPS\n", elapsedTime_gpu, gflops_gpu);
    printf("Naive GPU kernel execution time: %f ms, Performance: %f GFLOPS\n", elapsedTime_naive, gflops_naive);
    printf("Shared Memory GPU kernel execution time: %f ms, Performance: %f GFLOPS\n", elapsedTime_shared, gflops_shared);
    printf("CPU execution time: %f ms, Performance: %f GFLOPS\n", elapsedTime_cpu.count(), gflops_cpu);
    
    printf("Sample output (Optimized GPU):\n");
    for (int i = 0; i < 10; i++) printf("%d ", h_output_gpu[i]);
    printf("\n");
    
    printf("Sample output (Naive GPU):\n");
    for (int i = 0; i < 10; i++) printf("%d ", h_output_naive[i]);
    printf("\n");
    
    printf("Sample output (Shared Memory GPU):\n");
    for (int i = 0; i < 10; i++) printf("%d ", h_output_shared[i]);
    printf("\n");
    

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipEventDestroy(startNaive));
    CUDA_CHECK(hipEventDestroy(stopNaive));
    CUDA_CHECK(hipEventDestroy(startShared));
    CUDA_CHECK(hipEventDestroy(stopShared));
    
    free(h_image);
    free(h_output_gpu);
    free(h_output_naive);
    free(h_output_shared);
    free(h_output_cpu);
    
    return 0;
}
