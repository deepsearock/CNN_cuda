#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <fstream>
#include <sstream>

// Macro for error checking
#define CUDA_CHECK(call) {                                              \
    hipError_t err = call;                                             \
    if(err != hipSuccess) {                                             \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,  \
                hipGetErrorString(err));                               \
        exit(EXIT_FAILURE);                                             \
    }                                                                   \
}

//------------------------------------------------------
// 1. Optimized GPU Convolution Kernel (Texture & Shared Memory)
//------------------------------------------------------
texture<int, hipTextureType2D, hipReadModeElementType> texRef;

__global__ void convolution2DKernel(int *output, int imageWidth, int imageHeight, int maskWidth, int maskRadius) {
    extern __shared__ int sharedMem[];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;
    
    int sharedWidth = blockDim.x + 2 * maskRadius;
    int shared_x = tx + maskRadius;
    int shared_y = ty + maskRadius;
    
    // Load central data from texture memory
    if (row < imageHeight && col < imageWidth)
        sharedMem[shared_y * sharedWidth + shared_x] = tex2D(texRef, col, row);
    else
        sharedMem[shared_y * sharedWidth + shared_x] = 0;
    
    // Load halo regions (left, right, top, bottom, and corners)
    if (tx < maskRadius) {
        int halo_col = col - maskRadius;
        int value = (halo_col >= 0 && row < imageHeight) ? tex2D(texRef, halo_col, row) : 0;
        sharedMem[shared_y * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius) {
        int halo_col = col + maskRadius;
        int value = (halo_col < imageWidth && row < imageHeight) ? tex2D(texRef, halo_col, row) : 0;
        sharedMem[shared_y * sharedWidth + shared_x + maskRadius] = value;
    }
    if (ty < maskRadius) {
        int halo_row = row - maskRadius;
        int value = (halo_row >= 0 && col < imageWidth) ? tex2D(texRef, col, halo_row) : 0;
        sharedMem[ty * sharedWidth + shared_x] = value;
    }
    if (ty >= blockDim.y - maskRadius) {
        int halo_row = row + maskRadius;
        int value = (halo_row < imageHeight && col < imageWidth) ? tex2D(texRef, col, halo_row) : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + shared_x] = value;
    }
    // Corner halos:
    if (tx < maskRadius && ty < maskRadius) {
        int halo_col = col - maskRadius;
        int halo_row = row - maskRadius;
        int value = (halo_col >= 0 && halo_row >= 0) ? tex2D(texRef, halo_col, halo_row) : 0;
        sharedMem[ty * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius && ty < maskRadius) {
        int halo_col = col + maskRadius;
        int halo_row = row - maskRadius;
        int value = (halo_col < imageWidth && halo_row >= 0) ? tex2D(texRef, halo_col, halo_row) : 0;
        sharedMem[ty * sharedWidth + shared_x + maskRadius] = value;
    }
    if (tx < maskRadius && ty >= blockDim.y - maskRadius) {
        int halo_col = col - maskRadius;
        int halo_row = row + maskRadius;
        int value = (halo_col >= 0 && halo_row < imageHeight) ? tex2D(texRef, halo_col, halo_row) : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius && ty >= blockDim.y - maskRadius) {
        int halo_col = col + maskRadius;
        int halo_row = row + maskRadius;
        int value = (halo_col < imageWidth && halo_row < imageHeight) ? tex2D(texRef, halo_col, halo_row) : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + shared_x + maskRadius] = value;
    }
    
    __syncthreads();
    
    // Convolution operation using shared memory tile
    if (row < imageHeight && col < imageWidth) {
        int output_value = 0;
        for (int i = -maskRadius; i <= maskRadius; i++) {
            for (int j = -maskRadius; j <= maskRadius; j++) {
                int image_value = sharedMem[(shared_y + i) * sharedWidth + (shared_x + j)];
                output_value += image_value;
            }
        }
        output_value /= ((2 * maskRadius + 1) * (2 * maskRadius + 1));
        output[row * imageWidth + col] = output_value;
    }
}

//------------------------------------------------------
// 2. Naive GPU Convolution Kernel (Global Memory Only)
//------------------------------------------------------
__global__ void naiveConvolutionKernel(int *input, int *output, int imageWidth, int imageHeight, int maskWidth, int maskRadius) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < imageHeight && col < imageWidth) {
        int sum = 0;
        for (int i = -maskRadius; i <= maskRadius; i++) {
            for (int j = -maskRadius; j <= maskRadius; j++) {
                int r = row + i;
                int c = col + j;
                int value = 0;
                if (r >= 0 && r < imageHeight && c >= 0 && c < imageWidth)
                    value = input[r * imageWidth + c];
                sum += value;
            }
        }
        output[row * imageWidth + col] = sum / ((2 * maskRadius + 1) * (2 * maskRadius + 1));
    }
}

//------------------------------------------------------
// 3. Shared Memory GPU Convolution Kernel (Global Memory + Shared Memory)
//------------------------------------------------------
__global__ void sharedMemoryConvolutionKernel(int *input, int *output, int imageWidth, int imageHeight, int maskWidth, int maskRadius) {
    extern __shared__ int sharedMem[];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;
    
    int sharedWidth = blockDim.x + 2 * maskRadius;
    int shared_x = tx + maskRadius;
    int shared_y = ty + maskRadius;
    
    // Load central region from global memory into shared memory
    if (row < imageHeight && col < imageWidth)
        sharedMem[shared_y * sharedWidth + shared_x] = input[row * imageWidth + col];
    else
        sharedMem[shared_y * sharedWidth + shared_x] = 0;
    
    // Load halo regions from global memory.
    if (tx < maskRadius) {
        int halo_col = col - maskRadius;
        int value = (halo_col >= 0 && row < imageHeight) ? input[row * imageWidth + halo_col] : 0;
        sharedMem[shared_y * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius) {
        int halo_col = col + maskRadius;
        int value = (halo_col < imageWidth && row < imageHeight) ? input[row * imageWidth + halo_col] : 0;
        sharedMem[shared_y * sharedWidth + shared_x + maskRadius] = value;
    }
    if (ty < maskRadius) {
        int halo_row = row - maskRadius;
        int value = (halo_row >= 0 && col < imageWidth) ? input[halo_row * imageWidth + col] : 0;
        sharedMem[ty * sharedWidth + shared_x] = value;
    }
    if (ty >= blockDim.y - maskRadius) {
        int halo_row = row + maskRadius;
        int value = (halo_row < imageHeight && col < imageWidth) ? input[halo_row * imageWidth + col] : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + shared_x] = value;
    }
    // Corner halos:
    if (tx < maskRadius && ty < maskRadius) {
        int halo_col = col - maskRadius;
        int halo_row = row - maskRadius;
        int value = (halo_col >= 0 && halo_row >= 0) ? input[halo_row * imageWidth + halo_col] : 0;
        sharedMem[ty * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius && ty < maskRadius) {
        int halo_col = col + maskRadius;
        int halo_row = row - maskRadius;
        int value = (halo_col < imageWidth && halo_row >= 0) ? input[halo_row * imageWidth + halo_col] : 0;
        sharedMem[ty * sharedWidth + shared_x + maskRadius] = value;
    }
    if (tx < maskRadius && ty >= blockDim.y - maskRadius) {
        int halo_col = col - maskRadius;
        int halo_row = row + maskRadius;
        int value = (halo_col >= 0 && halo_row < imageHeight) ? input[halo_row * imageWidth + halo_col] : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + tx] = value;
    }
    if (tx >= blockDim.x - maskRadius && ty >= blockDim.y - maskRadius) {
        int halo_col = col + maskRadius;
        int halo_row = row + maskRadius;
        int value = (halo_col < imageWidth && halo_row < imageHeight) ? input[halo_row * imageWidth + halo_col] : 0;
        sharedMem[(shared_y + maskRadius) * sharedWidth + shared_x + maskRadius] = value;
    }
    
    __syncthreads();
    
    // Convolution operation using the shared memory tile
    if (row < imageHeight && col < imageWidth) {
        int sum = 0;
        for (int i = -maskRadius; i <= maskRadius; i++) {
            for (int j = -maskRadius; j <= maskRadius; j++) {
                sum += sharedMem[(shared_y + i) * sharedWidth + (shared_x + j)];
            }
        }
        output[row * imageWidth + col] = sum / ((2 * maskRadius + 1) * (2 * maskRadius + 1));
    }
}

//------------------------------------------------------
// 4. CPU Convolution Implementation
//------------------------------------------------------
void convolution2D_CPU(const int *h_image, int *h_output, int imageWidth, int imageHeight, int maskWidth) {
    int maskRadius = maskWidth / 2;
    for (int row = 0; row < imageHeight; row++) {
        for (int col = 0; col < imageWidth; col++) {
            int sum = 0;
            for (int i = -maskRadius; i <= maskRadius; i++) {
                for (int j = -maskRadius; j <= maskRadius; j++) {
                    int curRow = row + i;
                    int curCol = col + j;
                    int value = 0;
                    if (curRow >= 0 && curRow < imageHeight && curCol >= 0 && curCol < imageWidth)
                        value = h_image[curRow * imageWidth + curCol];
                    sum += value;
                }
            }
            h_output[row * imageWidth + col] = sum / ((2 * maskRadius + 1) * (2 * maskRadius + 1));
        }
    }
}

int main(int argc, char **argv) {
    // Default image dimensions: 512x512.
    int dimX = 512, dimY = 512;
    // Allow overriding image dimensions from command line.
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-i") == 0)
            dimX = atoi(argv[++i]);
        else if (strcmp(argv[i], "-j") == 0)
            dimY = atoi(argv[++i]);
    }
    
    int imageSize = dimX * dimY;
    
    // Allocate host memory for image and for kernel outputs.
    int *h_image         = (int *)malloc(sizeof(int) * imageSize);
    int *h_output_gpu    = (int *)malloc(sizeof(int) * imageSize); // Optimized GPU (texture+shared)
    int *h_output_naive  = (int *)malloc(sizeof(int) * imageSize); // Naive GPU (global only)
    int *h_output_shared = (int *)malloc(sizeof(int) * imageSize); // Shared Memory GPU
    int *h_output_cpu    = (int *)malloc(sizeof(int) * imageSize); // CPU result
    
    // Initialize image with random values between 0 and 15.
    for (int i = 0; i < imageSize; i++) {
        h_image[i] = rand() % 16;
    }
    
    // Define the mask sizes to test.
    int maskSizes[] = {4, 6, 8, 10, 12, 14, 16, 18, 20};
    int numMasks = sizeof(maskSizes) / sizeof(maskSizes[0]);
    
    // Fixed CUDA block and grid dimensions.
    dim3 blockDim(16, 16);
    dim3 gridDim((dimX + blockDim.x - 1) / blockDim.x, (dimY + blockDim.y - 1) / blockDim.y);
    
    // Build CSV file name to include image dimensions.
    std::ostringstream filename;
    filename << "results_" << dimX << "x" << dimY << ".csv";
    std::ofstream csvFile(filename.str());
    if (!csvFile.is_open()) {
        fprintf(stderr, "Error opening %s for writing.\n", filename.str().c_str());
        return EXIT_FAILURE;
    }
    // Write a comment line with image dimensions.
    csvFile << "# Image dimensions: " << dimX << " x " << dimY << "\n";
    // Write CSV header.
    csvFile << "MaskSize,OptimizedTime_ms,OptimizedGFLOPS,NaiveTime_ms,NaiveGFLOPS,SharedTime_ms,SharedGFLOPS,CPUTime_ms,CPUGFLOPS,Status\n";
    
    // Loop over each mask size.
    for (int m = 0; m < numMasks; m++) {
        int maskWidth = maskSizes[m];
        int maskRadius = maskWidth / 2;
        
        // Shared memory size based on the block and halo size.
        int sharedMemSize = (blockDim.x + 2 * maskRadius) * (blockDim.y + 2 * maskRadius) * sizeof(int);
        
        // ----------------------------
        // 1. Optimized GPU Convolution (Texture + Shared Memory)
        // ----------------------------
        hipArray *d_imageArray;
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int>();
        CUDA_CHECK(hipMallocArray(&d_imageArray, &channelDesc, dimX, dimY));
        CUDA_CHECK(hipMemcpy2DToArray(d_imageArray, 0, 0, h_image, dimX * sizeof(int),
                                       dimX * sizeof(int), dimY, hipMemcpyHostToDevice));
        // Bind texture.
        texRef.addressMode[0] = hipAddressModeClamp;
        texRef.addressMode[1] = hipAddressModeClamp;
        texRef.filterMode     = hipFilterModePoint;
        texRef.normalized     = false;
        CUDA_CHECK(hipBindTextureToArray(texRef, d_imageArray, channelDesc));
        
        int *d_output;
        CUDA_CHECK(hipMalloc(&d_output, sizeof(int) * imageSize));
        
        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));
        CUDA_CHECK(hipEventRecord(start, 0));
        convolution2DKernel<<<gridDim, blockDim, sharedMemSize>>>(d_output, dimX, dimY, maskWidth, maskRadius);
        CUDA_CHECK(hipEventRecord(stop, 0));
        CUDA_CHECK(hipEventSynchronize(stop));
        float elapsedTime_gpu;
        CUDA_CHECK(hipEventElapsedTime(&elapsedTime_gpu, start, stop));
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipMemcpy(h_output_gpu, d_output, sizeof(int) * imageSize, hipMemcpyDeviceToHost));
        
        CUDA_CHECK(hipUnbindTexture(texRef));
        CUDA_CHECK(hipFreeArray(d_imageArray));
        CUDA_CHECK(hipFree(d_output));
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
        
        // ----------------------------
        // 2. Naive GPU Convolution (Global Memory Only)
        // ----------------------------
        int *d_input, *d_output_naive;
        CUDA_CHECK(hipMalloc(&d_input, sizeof(int) * imageSize));
        CUDA_CHECK(hipMalloc(&d_output_naive, sizeof(int) * imageSize));
        CUDA_CHECK(hipMemcpy(d_input, h_image, sizeof(int) * imageSize, hipMemcpyHostToDevice));
        
        hipEvent_t startNaive, stopNaive;
        CUDA_CHECK(hipEventCreate(&startNaive));
        CUDA_CHECK(hipEventCreate(&stopNaive));
        CUDA_CHECK(hipEventRecord(startNaive, 0));
        naiveConvolutionKernel<<<gridDim, blockDim>>>(d_input, d_output_naive, dimX, dimY, maskWidth, maskRadius);
        CUDA_CHECK(hipEventRecord(stopNaive, 0));
        CUDA_CHECK(hipEventSynchronize(stopNaive));
        float elapsedTime_naive;
        CUDA_CHECK(hipEventElapsedTime(&elapsedTime_naive, startNaive, stopNaive));
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipMemcpy(h_output_naive, d_output_naive, sizeof(int) * imageSize, hipMemcpyDeviceToHost));
        
        CUDA_CHECK(hipFree(d_input));
        CUDA_CHECK(hipFree(d_output_naive));
        CUDA_CHECK(hipEventDestroy(startNaive));
        CUDA_CHECK(hipEventDestroy(stopNaive));
        
        // ----------------------------
        // 3. Shared Memory GPU Convolution (Global Memory + Shared Memory)
        // ----------------------------
        int *d_input_shared, *d_output_shared;
        CUDA_CHECK(hipMalloc(&d_input_shared, sizeof(int) * imageSize));
        CUDA_CHECK(hipMalloc(&d_output_shared, sizeof(int) * imageSize));
        CUDA_CHECK(hipMemcpy(d_input_shared, h_image, sizeof(int) * imageSize, hipMemcpyHostToDevice));
        
        hipEvent_t startShared, stopShared;
        CUDA_CHECK(hipEventCreate(&startShared));
        CUDA_CHECK(hipEventCreate(&stopShared));
        CUDA_CHECK(hipEventRecord(startShared, 0));
        sharedMemoryConvolutionKernel<<<gridDim, blockDim, sharedMemSize>>>(d_input_shared, d_output_shared, dimX, dimY, maskWidth, maskRadius);
        CUDA_CHECK(hipEventRecord(stopShared, 0));
        CUDA_CHECK(hipEventSynchronize(stopShared));
        float elapsedTime_shared;
        CUDA_CHECK(hipEventElapsedTime(&elapsedTime_shared, startShared, stopShared));
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipMemcpy(h_output_shared, d_output_shared, sizeof(int) * imageSize, hipMemcpyDeviceToHost));
        
        CUDA_CHECK(hipFree(d_input_shared));
        CUDA_CHECK(hipFree(d_output_shared));
        CUDA_CHECK(hipEventDestroy(startShared));
        CUDA_CHECK(hipEventDestroy(stopShared));
        
        // ----------------------------
        // 4. CPU Convolution
        // ----------------------------
        auto cpu_start = std::chrono::high_resolution_clock::now();
        convolution2D_CPU(h_image, h_output_cpu, dimX, dimY, maskWidth);
        auto cpu_end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> elapsedTime_cpu = cpu_end - cpu_start;
        
        // ----------------------------
        // 5. Compare Results (all kernels vs CPU)
        // ----------------------------
        const double epsilon = 1e-5;
        bool correct = true;
        for (int i = 0; i < imageSize; i++) {
            if (abs(h_output_gpu[i] - h_output_cpu[i]) > epsilon ||
                abs(h_output_naive[i] - h_output_cpu[i]) > epsilon ||
                abs(h_output_shared[i] - h_output_cpu[i]) > epsilon) {
                correct = false;
                break;
            }
        }
        std::string status = correct ? "Correct" : "Mismatch";
        
        // ----------------------------
        // 6. Performance Calculations
        // ----------------------------
        // Note: Using opsPerPixel = (maskWidth * maskWidth) + 1 as in original code.
        double opsPerPixel = (maskWidth * maskWidth) + 1;
        double totalOps = imageSize * opsPerPixel;
        
        double seconds_gpu    = elapsedTime_gpu / 1000.0;
        double gflops_gpu     = (totalOps / seconds_gpu) / 1e9;
        
        double seconds_naive  = elapsedTime_naive / 1000.0;
        double gflops_naive   = (totalOps / seconds_naive) / 1e9;
        
        double seconds_shared = elapsedTime_shared / 1000.0;
        double gflops_shared  = (totalOps / seconds_shared) / 1e9;
        
        double seconds_cpu    = elapsedTime_cpu.count() / 1000.0;
        double gflops_cpu     = (totalOps / seconds_cpu) / 1e9;
        
        // ----------------------------
        // 7. Write results to CSV file.
        // ----------------------------
        csvFile << maskWidth << ","
                << elapsedTime_gpu << "," << gflops_gpu << ","
                << elapsedTime_naive << "," << gflops_naive << ","
                << elapsedTime_shared << "," << gflops_shared << ","
                << elapsedTime_cpu.count() << "," << gflops_cpu << ","
                << status << "\n";
    }
    
    csvFile.close();
    printf("All kernels produced correct results. Results written to %s\n", filename.str().c_str());
    
    // Clean up host memory.
    free(h_image);
    free(h_output_gpu);
    free(h_output_naive);
    free(h_output_shared);
    free(h_output_cpu);
    
    return 0;
}
