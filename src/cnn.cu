#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <getopt.h>
#include "cnn_naive.cuh"  // Include the header file with the naive kernel
#include "utils.cuh"  // Include utils for performance measurement

// Function to launch the convolution kernel
PerformanceMetrics cnn_naive(float *h_input, float *h_output, float *h_mask, int dimX, int dimY, int dimK) {
    float *d_input, *d_output, *d_mask;
    size_t img_size = dimX * dimY * sizeof(float);
    size_t mask_size = dimK * dimK * sizeof(float);
    
    // Allocate device memory
    hipMalloc((void**)&d_input, img_size);
    hipMalloc((void**)&d_output, img_size);
    hipMalloc((void**)&d_mask, mask_size);
    
    // Copy data to device
    hipMemcpy(d_input, h_input, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, mask_size, hipMemcpyHostToDevice);
    
    // Set grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((dimX + blockDim.x - 1) / blockDim.x, 
                 (dimY + blockDim.y - 1) / blockDim.y);
    
    // Measure performance using the naive kernel
    PerformanceMetrics metrics = measurePerformance((void*)naiveConvolution2D, false,
                                                   d_input, d_mask, d_output, 
                                                   dimX, dimY, dimK, dimK,
                                                   gridDim, blockDim);
    
    printf("Naive Convolution Performance: %f ms, %f GFLOPS\n", 
           metrics.executionTime, metrics.gflops);

    // Copy result back to host
    hipMemcpy(h_output, d_output, img_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mask);
    
    return metrics;
}

// Function to launch the optimized convolution kernel using texture memory
PerformanceMetrics cnn_optimized(float *h_input, float *h_output, float *h_mask, int dimX, int dimY, int dimK) {
    float *d_input, *d_output, *d_mask;
    size_t img_size = dimX * dimY * sizeof(float);
    size_t mask_size = dimK * dimK * sizeof(float);
    
    // Allocate device memory
    hipMalloc((void**)&d_input, img_size);
    hipMalloc((void**)&d_output, img_size);
    hipMalloc((void**)&d_mask, mask_size);
    
    // Copy data to device
    hipMemcpy(d_input, h_input, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, mask_size, hipMemcpyHostToDevice);
    
    // Set up texture reference for input image
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, dimX, dimY);
    hipMemcpyToArray(cuArray, 0, 0, h_input, img_size, hipMemcpyHostToDevice);
    
    // Bind texture reference to the CUDA array
    texRef.addressMode[0] = hipAddressModeClamp;
    texRef.addressMode[1] = hipAddressModeClamp;
    texRef.filterMode = hipFilterModePoint;
    texRef.normalized = false;
    hipBindTextureToArray(texRef, cuArray, channelDesc);
    
    // Set grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((dimX + blockDim.x - 1) / blockDim.x, 
                 (dimY + blockDim.y - 1) / blockDim.y);
    
    // Measure performance using the optimized kernel
    PerformanceMetrics metrics = measurePerformance((void*)optimizedConvolution2D, true,
                                                   d_input, d_mask, d_output, 
                                                   dimX, dimY, dimK, dimK,
                                                   gridDim, blockDim);
    
    printf("Optimized Convolution Performance: %f ms, %f GFLOPS\n", 
           metrics.executionTime, metrics.gflops);

    // Copy result back to host
    hipMemcpy(h_output, d_output, img_size, hipMemcpyDeviceToHost);

    // Unbind texture
    hipUnbindTexture(texRef);
    
    // Free device memory
    hipFreeArray(cuArray);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mask);
    
    return metrics;
}