#include "../include/cnn_launcher.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Define the texture reference for optimized kernel
texture<float, hipTextureType2D, hipReadModeElementType> texRef;

// CPU performance measurement function
PerformanceMetrics cnn_cpu(float *h_input, float *h_output, float *h_mask, 
                           int dimX, int dimY, int dimK) {
    PerformanceMetrics metrics;
    
    // Calculate total number of operations (multiply-adds)
    long long totalOps = static_cast<long long>(dimX) * dimY * (2 * dimK * dimK - 1);
    
    // Use CUDA events for consistent timing with GPU functions
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    // Run CPU convolution
    cpuConvolution2D(h_input, h_mask, h_output, dimX, dimY, dimK, dimK);
    
    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Calculate performance metrics
    metrics.executionTime = milliseconds;
    metrics.gflops = (totalOps / (milliseconds * 1.0e6));
    
    printf("CPU Convolution Performance: %f ms, %f GFLOPS\n", 
           metrics.executionTime, metrics.gflops);
    
    // Clean up events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return metrics;
}

// Function to launch the naive convolution kernel
PerformanceMetrics cnn_naive(float *h_input, float *h_output, float *h_mask, 
                             int dimX, int dimY, int dimK) {
    float *d_input, *d_output, *d_mask;
    size_t img_size = dimX * dimY * sizeof(float);
    size_t mask_size = dimK * dimK * sizeof(float);
    
    // Allocate device memory
    hipMalloc((void**)&d_input, img_size);
    hipMalloc((void**)&d_output, img_size);
    hipMalloc((void**)&d_mask, mask_size);
    
    // Copy data to device
    hipMemcpy(d_input, h_input, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, mask_size, hipMemcpyHostToDevice);
    
    // Set grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((dimX + blockDim.x - 1) / blockDim.x, 
                 (dimY + blockDim.y - 1) / blockDim.y);
    
    // Measure performance using the naive kernel
    PerformanceMetrics metrics = measurePerformance((void*)naiveConvolution2D, KernelType::NAIVE,
                                                      d_input, d_mask, d_output, 
                                                      dimX, dimY, dimK, dimK,
                                                      gridDim, blockDim);
    
    printf("Naive Convolution Performance: %f ms, %f GFLOPS\n", 
           metrics.executionTime, metrics.gflops);

    // Copy result back to host
    hipMemcpy(h_output, d_output, img_size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mask);
    
    return metrics;
}

// Function to launch the optimized convolution kernel using texture memory
PerformanceMetrics cnn_optimized(float *h_input, float *h_output, float *h_mask, 
                                 int dimX, int dimY, int dimK) {
    float *d_output, *d_mask;
    size_t img_size = dimX * dimY * sizeof(float);
    size_t mask_size = dimK * dimK * sizeof(float);
    
    // Allocate device memory for output and mask
    hipMalloc((void**)&d_output, img_size);
    hipMalloc((void**)&d_mask, mask_size);
    
    // Copy mask data to device
    hipMemcpy(d_mask, h_mask, mask_size, hipMemcpyHostToDevice);
    
    // Set up texture reference for input image
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, dimX, dimY);
    hipMemcpyToArray(cuArray, 0, 0, h_input, img_size, hipMemcpyHostToDevice);
    
    // Bind texture reference to the CUDA array
    texRef.addressMode[0] = hipAddressModeClamp;
    texRef.addressMode[1] = hipAddressModeClamp;
    texRef.filterMode = hipFilterModePoint;
    texRef.normalized = false;
    hipBindTextureToArray(texRef, cuArray, channelDesc);
    
    // Set grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((dimX + blockDim.x - 1) / blockDim.x, 
                 (dimY + blockDim.y - 1) / blockDim.y);
    
    // Measure performance using the optimized kernel
    PerformanceMetrics metrics = measurePerformance((void*)optimizedConvolution2D, KernelType::OPTIMIZED,
                                                      nullptr, d_mask, d_output, 
                                                      dimX, dimY, dimK, dimK,
                                                      gridDim, blockDim);
    
    // Copy result back to host
    hipMemcpy(h_output, d_output, img_size, hipMemcpyDeviceToHost);

    // Unbind texture and free CUDA array
    hipUnbindTexture(texRef);
    hipFreeArray(cuArray);
    
    // Free device memory
    hipFree(d_output);
    hipFree(d_mask);
    
    return metrics;
}

// Function to launch the vectorized convolution kernel
PerformanceMetrics cnn_vectorized(float *h_input, float *h_output, float *h_mask, 
                                  int dimX, int dimY, int dimK) {
    float *d_input, *d_output, *d_mask;
    size_t img_size = dimX * dimY * sizeof(float);
    size_t mask_size = dimK * dimK * sizeof(float);
    
    // Allocate device memory
    hipMalloc((void**)&d_input, img_size);
    hipMalloc((void**)&d_output, img_size);
    hipMalloc((void**)&d_mask, mask_size);
    
    // Copy data to device
    hipMemcpy(d_input, h_input, img_size, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, mask_size, hipMemcpyHostToDevice);
    
    // Set grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((dimX + blockDim.x - 1) / blockDim.x, 
                 (dimY + blockDim.y - 1) / blockDim.y);
    
    // Measure performance using the vectorized kernel
    PerformanceMetrics metrics = measurePerformance((void*)vectorizedConvolution2D, KernelType::VECTORIZED,
                                                      d_input, d_mask, d_output, 
                                                      dimX, dimY, dimK, dimK,
                                                      gridDim, blockDim);
    
    // Copy result back to host
    hipMemcpy(h_output, d_output, img_size, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mask);
    
    return metrics;
}
